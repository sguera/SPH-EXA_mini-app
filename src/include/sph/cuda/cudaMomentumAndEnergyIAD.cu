#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <algorithm>

#include "../kernels.hpp"
#include "sph.cuh"
#include "utils.cuh"

namespace sphexa
{
namespace sph
{
namespace cuda
{
namespace kernels
{
const double gradh_i = 1.0;
const double gradh_j = 1.0;

template <typename T>
__global__ void computeMomentumAndEnergyIAD(const int n, const T sincIndex, const T K, const int ngmax, const BBox<T> *bbox,
                                            const int *clist, const int *neighbors, const int *neighborsCount, const T *x, const T *y,
                                            const T *z, const T *vx, const T *vy, const T *vz, const T *h, const T *m, const T *ro,
                                            const T *p, const T *c, const T *c11, const T *c12, const T *c13, const T *c22, const T *c23,
                                            const T *c33, T *grad_P_x, T *grad_P_y, T *grad_P_z, T *du)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= n) return;

    const int i = clist[tid];
    const int nn = neighborsCount[tid];

    T momentum_x = 0.0, momentum_y = 0.0, momentum_z = 0.0, energy = 0.0, energyAV = 0.0;
    for (int pj = 0; pj < nn; ++pj)
    {
        const int j = neighbors[tid * ngmax + pj];

        T r_ijx = (x[i] - x[j]);
        T r_ijy = (y[i] - y[j]);
        T r_ijz = (z[i] - z[j]);

        T r_jix = (x[j] - x[i]);
        T r_jiy = (y[j] - y[i]);
        T r_jiz = (z[j] - z[i]);

        applyPBC(*bbox, 2.0 * h[i], r_ijx, r_ijy, r_ijz);
        applyPBC(*bbox, 2.0 * h[i], r_jix, r_jiy, r_jiz);

        const T dist = std::sqrt(r_ijx * r_ijx + r_ijy * r_ijy + r_ijz * r_ijz);

        const T v_ijx = (vx[i] - vx[j]);
        const T v_ijy = (vy[i] - vy[j]);
        const T v_ijz = (vz[i] - vz[j]);

        const T v1 = dist / h[i];
        const T v2 = dist / h[j];

        const T rv = r_ijx * v_ijx + r_ijy * v_ijy + r_ijz * v_ijz;

        const T w1 = K * math_namespace::pow(wharmonic(v1), (int)sincIndex);
        const T w2 = K * math_namespace::pow(wharmonic(v2), (int)sincIndex);

        const T W1 = w1 / (h[i] * h[i] * h[i]);
        const T W2 = w2 / (h[j] * h[j] * h[j]);

        const T kern11_i = c11[i] * r_jix;
        const T kern12_i = c12[i] * r_jiy;
        const T kern13_i = c13[i] * r_jiz;
        const T kern21_i = c12[i] * r_jix;
        const T kern22_i = c22[i] * r_jiy;
        const T kern23_i = c23[i] * r_jiz;
        const T kern31_i = c13[i] * r_jix;
        const T kern32_i = c23[i] * r_jiy;
        const T kern33_i = c33[i] * r_jiz;

        const T kern11_j = c11[j] * r_jix;
        const T kern12_j = c12[j] * r_jiy;
        const T kern13_j = c13[j] * r_jiz;
        const T kern21_j = c12[j] * r_jix;
        const T kern22_j = c22[j] * r_jiy;
        const T kern23_j = c23[j] * r_jiz;
        const T kern31_j = c13[j] * r_jix;
        const T kern32_j = c23[j] * r_jiy;
        const T kern33_j = c33[j] * r_jiz;

        const T termA1_i = (kern11_i + kern12_i + kern13_i) * W1;
        const T termA2_i = (kern21_i + kern22_i + kern23_i) * W1;
        const T termA3_i = (kern31_i + kern32_i + kern33_i) * W1;

        const T termA1_j = (kern11_j + kern12_j + kern13_j) * W2;
        const T termA2_j = (kern21_j + kern22_j + kern23_j) * W2;
        const T termA3_j = (kern31_j + kern32_j + kern33_j) * W2;

        const T pro_i = p[i] / (gradh_i * ro[i] * ro[i]);
        const T pro_j = p[j] / (gradh_j * ro[j] * ro[j]);

        const T r_square = dist * dist;
        const T viscosity_ij = artificial_viscosity(ro[i], ro[j], h[i], h[j], c[i], c[j], rv, r_square);

        const T grad_Px_AV = 0.5 * (m[i] / ro[i] * viscosity_ij * termA1_i + m[j] / ro[j] * viscosity_ij * termA1_j);
        const T grad_Py_AV = 0.5 * (m[i] / ro[i] * viscosity_ij * termA2_i + m[j] / ro[j] * viscosity_ij * termA2_j);
        const T grad_Pz_AV = 0.5 * (m[i] / ro[i] * viscosity_ij * termA3_i + m[j] / ro[j] * viscosity_ij * termA3_j);

        momentum_x += m[j] * (pro_i * termA1_i + pro_j * termA1_j) + grad_Px_AV;
        momentum_y += m[j] * (pro_i * termA2_i + pro_j * termA2_j) + grad_Py_AV;
        momentum_z += m[j] * (pro_i * termA3_i + pro_j * termA3_j) + grad_Pz_AV;

        energy += m[j] * 2.0 * pro_i * (v_ijx * termA1_i + v_ijy * termA2_i + v_ijz * termA3_i);
        energyAV += grad_Px_AV * v_ijx + grad_Py_AV * v_ijy + grad_Pz_AV * v_ijz;
    }

    du[tid] = 0.5 * (energy + energyAV);
    grad_P_x[tid] = momentum_x;
    grad_P_y[tid] = momentum_y;
    grad_P_z[tid] = momentum_z;
}
} // namespace kernels

template void computeMomentumAndEnergyIAD<double, ParticlesData<double>>(const std::vector<ParticleIdxChunk> &chunksToCompute, ParticlesData<double> &d);

template <typename T, class Dataset>
void computeMomentumAndEnergyIAD(const std::vector<ParticleIdxChunk> &chunksToCompute, Dataset &d)
{
    const size_t np = d.x.size();
    const size_t size_np_T = np * sizeof(T);

    const auto largestChunkSize =
        std::max_element(chunksToCompute.cbegin(), chunksToCompute.cend(),
                         [](const std::vector<int> &lhs, const std::vector<int> &rhs) { return lhs.size() < rhs.size(); })
            ->size();

    const size_t size_largerNeighborsChunk_int = largestChunkSize * d.ngmax * sizeof(int);
    const size_t size_largerNChunk_int = largestChunkSize * sizeof(int);
    const size_t size_largerNChunk_T = largestChunkSize * sizeof(T);
    const size_t size_bbox = sizeof(BBox<T>);

    // const size_t size_bbox = sizeof(BBox<T>);
    // const size_t size_np_T = np * sizeof(T);
    // const size_t size_n_int = n * sizeof(int);
    // const size_t size_n_T = n * sizeof(T);
    // const size_t size_allNeighbors = allNeighbors * sizeof(int);

    int *d_clist, *d_neighbors, *d_neighborsCount;
    T *d_x, *d_y, *d_z, *d_vx, *d_vy, *d_vz, *d_m, *d_h, *d_ro, *d_p, *d_c, *d_c11, *d_c12, *d_c13, *d_c22, *d_c23, *d_c33;
    BBox<T> *d_bbox;
    T *d_grad_P_x, *d_grad_P_y, *d_grad_P_z, *d_du;

    // input data
    CHECK_CUDA_ERR(
        utils::hipMalloc(size_np_T, d_x, d_y, d_z, d_vx, d_vy, d_vz, d_h, d_m, d_ro, d_p, d_c, d_c11, d_c12, d_c13, d_c22, d_c23, d_c33));
    CHECK_CUDA_ERR(utils::hipMalloc(size_bbox, d_bbox));
    CHECK_CUDA_ERR(utils::hipMalloc(size_largerNChunk_int, d_clist, d_neighborsCount));
    CHECK_CUDA_ERR(utils::hipMalloc(size_largerNeighborsChunk_int, d_neighbors));

    // oputput data
    CHECK_CUDA_ERR(utils::hipMalloc(size_largerNChunk_T, d_grad_P_x, d_grad_P_y, d_grad_P_z, d_du));

    CHECK_CUDA_ERR(hipMemcpy(d_x, d.x.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_y, d.y.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_z, d.z.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_vx, d.vx.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_vy, d.vy.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_vz, d.vz.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_h, d.h.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_m, d.m.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_ro, d.ro.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_p, d.p.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_c, d.c.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_c11, d.c11.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_c12, d.c12.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_c13, d.c13.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_c22, d.c22.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_c23, d.c23.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_c33, d.c33.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_bbox, &d.bbox, size_bbox, hipMemcpyHostToDevice));

    for (const auto &clist : chunksToCompute)
    {
        const size_t n = clist.size();
        const size_t size_n_T = n * sizeof(T);
        const size_t size_n_int = n * sizeof(int);
        const size_t size_nNeighbors = n * d.ngmax * sizeof(int);

        const size_t neighborsOffset = clist.front() * d.ngmax;
        const int *neighbors = d.neighbors.data() + neighborsOffset;

        const size_t neighborsCountOffset = clist.front();
        const int *neighborsCount = d.neighborsCount.data() + neighborsCountOffset;

        CHECK_CUDA_ERR(hipMemcpy(d_clist, clist.data(), size_n_int, hipMemcpyHostToDevice));
        CHECK_CUDA_ERR(hipMemcpy(d_neighbors, neighbors, size_nNeighbors, hipMemcpyHostToDevice));
        CHECK_CUDA_ERR(hipMemcpy(d_neighborsCount, neighborsCount, size_n_int, hipMemcpyHostToDevice));

        const int threadsPerBlock = 256;
        const int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

        kernels::computeMomentumAndEnergyIAD<<<blocksPerGrid, threadsPerBlock>>>(
            n, d.sincIndex, d.K, d.ngmax, d_bbox, d_clist, d_neighbors, d_neighborsCount, d_x, d_y, d_z, d_vx, d_vy, d_vz, d_h, d_m, d_ro,
            d_p, d_c, d_c11, d_c12, d_c13, d_c22, d_c23, d_c33, d_grad_P_x, d_grad_P_y, d_grad_P_z, d_du);

        CHECK_CUDA_ERR(hipGetLastError());

        CHECK_CUDA_ERR(hipMemcpy(d.grad_P_x.data() + clist.front(), d_grad_P_x, size_n_T, hipMemcpyDeviceToHost));
        CHECK_CUDA_ERR(hipMemcpy(d.grad_P_y.data() + clist.front(), d_grad_P_y, size_n_T, hipMemcpyDeviceToHost));
        CHECK_CUDA_ERR(hipMemcpy(d.grad_P_z.data() + clist.front(), d_grad_P_z, size_n_T, hipMemcpyDeviceToHost));
        CHECK_CUDA_ERR(hipMemcpy(d.du.data() + clist.front(), d_du, size_n_T, hipMemcpyDeviceToHost));
    }

    CHECK_CUDA_ERR(utils::hipFree(d_clist, d_neighborsCount, d_neighbors, d_bbox, d_x, d_y, d_z, d_vx, d_vy, d_vz, d_h, d_m, d_ro, d_p,
                                   d_c, d_c11, d_c12, d_c13, d_c22, d_c23, d_c33, d_grad_P_x, d_grad_P_y, d_grad_P_z, d_du));
}

} // namespace cuda
} // namespace sph
} // namespace sphexa
