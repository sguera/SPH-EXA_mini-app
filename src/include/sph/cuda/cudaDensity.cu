#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <algorithm>
#include <omp.h>

#include "sph.cuh"
#include "BBox.hpp"
#include "ParticlesData.hpp"
#include "cudaUtils.cuh"
#include "../kernels.hpp"
#include "../lookupTables.hpp"

namespace sphexa
{
namespace sph
{
namespace cuda
{
namespace kernels
{
template <typename T>
__global__ void density(const int n, const T sincIndex, const T K, const int ngmax, const BBox<T> *bbox, const int *clist,
                        const int *neighbors, const int *neighborsCount, const T *x, const T *y, const T *z, const T *h, const T *m, T *ro)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid >= n) return;

    const int i = clist[tid];
    const int nn = neighborsCount[tid];

    T roloc = 0.0;

    for (int pj = 0; pj < nn; ++pj)
    {
        const int j = neighbors[tid * ngmax + pj];
        const T dist = distancePBC(*bbox, h[i], x[i], y[i], z[i], x[j], y[j], z[j]);
        const T vloc = dist / h[i];
        const T w = K * math_namespace::pow(wharmonic(vloc), (int)sincIndex);
        const T value = w / (h[i] * h[i] * h[i]);
        roloc += value * m[j];
    }

    ro[i] = roloc + m[i] * K / (h[i] * h[i] * h[i]);
}
} // namespace kernels

template <typename T, class ParticleData>
void computeDensity(const std::vector<Task> &taskList, ParticleData &d)
{
    const size_t np = d.x.size();
    const size_t size_np_T = np * sizeof(T);

    const auto largestChunkSize = std::max_element(taskList.cbegin(), taskList.cend(),
                                                   [](const Task &lhs, const Task &rhs) { return lhs.clist.size() < rhs.clist.size(); })
                                      ->clist.size();

    const size_t size_largerNeighborsChunk_int = largestChunkSize * Task::ngmax * sizeof(int);
    const size_t size_largerNChunk_int = largestChunkSize * sizeof(int);
    const size_t size_bbox = sizeof(BBox<T>);

    // device pointers - d_ prefix stands for device
    int ts = taskList.size();
    int *d_clist[ts], *d_neighbors[ts], *d_neighborsCount[ts];
    T *d_x, *d_y, *d_z, *d_m, *d_h;
    T *d_ro;
    BBox<T> *d_bbox;

    // input data
    CHECK_CUDA_ERR(utils::hipMalloc(size_np_T, d_x, d_y, d_z, d_h, d_m, d_ro));
    CHECK_CUDA_ERR(utils::hipMalloc(size_bbox, d_bbox));
    // CHECK_CUDA_ERR(utils::hipMalloc(size_largerNChunk_int, d_clist, d_neighborsCount));
    // CHECK_CUDA_ERR(utils::hipMalloc(size_largerNeighborsChunk_int, d_neighbors));

    CHECK_CUDA_ERR(hipMemcpy(d_x, d.x.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_y, d.y.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_z, d.z.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_h, d.h.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_m, d.m.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_bbox, &d.bbox, size_bbox, hipMemcpyHostToDevice));

    const int nStreams = taskList.size();
    hipStream_t streams[nStreams];
    int i = 0;
    for (int i = 0; i < nStreams; i++)
        hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);

    // #pragma omp parallel
    // #pragma omp single
    for (const auto &t : taskList)
    {
        // #pragma omp task
        {
            const size_t n = t.clist.size();
            const size_t size_n_int = n * sizeof(int);
            const size_t size_nNeighbors = n * Task::ngmax * sizeof(int);
            CHECK_CUDA_ERR(utils::hipMalloc(size_largerNChunk_int, d_clist[i], d_neighborsCount[i]));
            CHECK_CUDA_ERR(utils::hipMalloc(size_largerNeighborsChunk_int, d_neighbors[i]));

            // CHECK_CUDA_ERR(hipMemcpy(d_clist, t.clist.data(), size_n_int, hipMemcpyHostToDevice));
            // CHECK_CUDA_ERR(hipMemcpy(d_neighbors, t.neighbors.data(), size_nNeighbors, hipMemcpyHostToDevice));
            // CHECK_CUDA_ERR(hipMemcpy(d_neighborsCount, t.neighborsCount.data(), size_n_int, hipMemcpyHostToDevice));

            // if (i != 0) hipStreamSynchronize(streams[i - 1]);

            CHECK_CUDA_ERR(hipMemcpyAsync(d_clist[i], t.clist.data(), size_n_int, hipMemcpyHostToDevice, streams[i]));
            CHECK_CUDA_ERR(hipMemcpyAsync(d_neighbors[i], t.neighbors.data(), size_nNeighbors, hipMemcpyHostToDevice, streams[i]));
            CHECK_CUDA_ERR(hipMemcpyAsync(d_neighborsCount[i], t.neighborsCount.data(), size_n_int, hipMemcpyHostToDevice, streams[i]));

            const int threadsPerBlock = 256;
            const int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

            // printf("CUDA Density kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

            // kernels::density<<<blocksPerGrid, threadsPerBlock>>>(n, d.sincIndex, d.K, t.ngmax, d_bbox, d_clist, d_neighbors,
            //                                                      d_neighborsCount, d_x, d_y, d_z, d_h, d_m, d_ro);
            kernels::density<<<blocksPerGrid, threadsPerBlock, 0, streams[i]>>>(
                n, d.sincIndex, d.K, t.ngmax, d_bbox, d_clist[i], d_neighbors[i], d_neighborsCount[i], d_x, d_y, d_z, d_h, d_m, d_ro);
            CHECK_CUDA_ERR(hipGetLastError());

            ++i;
        }
    }
    // #pragma omp taskwait

    for (int i = 0; i < nStreams; i++)
    {
        hipStreamSynchronize(streams[i]);
        CHECK_CUDA_ERR(utils::hipFree(d_clist[i], d_neighbors[i], d_neighborsCount[i]));
    }

    CHECK_CUDA_ERR(hipMemcpy(d.ro.data(), d_ro, size_np_T, hipMemcpyDeviceToHost));

    CHECK_CUDA_ERR(utils::hipFree(d_x, d_y, d_z, d_h, d_m, d_bbox, d_ro));
}

template <typename T, class ParticleData>
void computeDensity(const Task &task, ParticleData &d)
{
    const size_t np = d.x.size();
    const size_t size_np_T = np * sizeof(T);

    const auto largestChunkSize = task.clist.size();
    // std::max_element(taskList.cbegin(), taskList.cend(),
    //                                                [](const Task &lhs, const Task &rhs) { return lhs.clist.size() < rhs.clist.size(); })
    //                                   ->clist.size();

    const size_t size_largerNeighborsChunk_int = largestChunkSize * Task::ngmax * sizeof(int);
    const size_t size_largerNChunk_int = largestChunkSize * sizeof(int);
    const size_t size_bbox = sizeof(BBox<T>);

    // device pointers - d_ prefix stands for device
    int *d_clist, *d_neighbors, *d_neighborsCount;
    // T *d_x, *d_y, *d_z, *d_m, *d_h;
    // T *d_ro;
    // BBox<T> *d_bbox;

    // input data
    // CHECK_CUDA_ERR(utils::hipMalloc(size_np_T, d_x, d_y, d_z, d_h, d_m, d_ro));
    // CHECK_CUDA_ERR(utils::hipMalloc(size_bbox, d_bbox));
    CHECK_CUDA_ERR(utils::hipMalloc(size_largerNChunk_int, d_clist, d_neighborsCount));
    CHECK_CUDA_ERR(utils::hipMalloc(size_largerNeighborsChunk_int, d_neighbors));

    // CHECK_CUDA_ERR(hipMemcpyAsync(d_x, d.x.data(), size_np_T, hipMemcpyHostToDevice));
    // CHECK_CUDA_ERR(hipMemcpyAsync(d_y, d.y.data(), size_np_T, hipMemcpyHostToDevice));
    // CHECK_CUDA_ERR(hipMemcpyAsync(d_z, d.z.data(), size_np_T, hipMemcpyHostToDevice));
    // CHECK_CUDA_ERR(hipMemcpyAsync(d_h, d.h.data(), size_np_T, hipMemcpyHostToDevice));
    // CHECK_CUDA_ERR(hipMemcpyAsync(d_m, d.m.data(), size_np_T, hipMemcpyHostToDevice));
    // CHECK_CUDA_ERR(hipMemcpyAsync(d_bbox, &d.bbox, size_bbox, hipMemcpyHostToDevice));

    // const int nStreams = taskList.size();
    // hipStream_t streams[nStreams];
    // int i = 0;
    // for (int i = 0; i < nStreams; i++)
    //     hipStreamCreate(&streams[i]);

    const size_t n = task.clist.size();
    const size_t size_n_int = n * sizeof(int);
    const size_t size_nNeighbors = n * Task::ngmax * sizeof(int);

    // CHECK_CUDA_ERR(hipMemcpy(d_clist, task.clist.data(), size_n_int, hipMemcpyHostToDevice));
    // CHECK_CUDA_ERR(hipMemcpy(d_neighbors, task.neighbors.data(), size_nNeighbors, hipMemcpyHostToDevice));
    // CHECK_CUDA_ERR(hipMemcpy(d_neighborsCount, task.neighborsCount.data(), size_n_int, hipMemcpyHostToDevice));

    CHECK_CUDA_ERR(hipMemcpyAsync(d_clist, task.clist.data(), size_n_int, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpyAsync(d_neighbors, task.neighbors.data(), size_nNeighbors, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpyAsync(d_neighborsCount, task.neighborsCount.data(), size_n_int, hipMemcpyHostToDevice));

    const int threadsPerBlock = 256;
    const int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    // printf("CUDA Density kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    kernels::density<<<blocksPerGrid, threadsPerBlock>>>(n, d.sincIndex, d.K, task.ngmax, d.d_bbox, d_clist, d_neighbors, d_neighborsCount,
                                                         d.d_x, d.d_y, d.d_z, d.d_h, d.d_m, d.d_ro);
    CHECK_CUDA_ERR(hipGetLastError());

    // CHECK_CUDA_ERR(hipMemcpyAsync(d.ro.data() + task.clist.front(), d_ro + task.clist.front(), task.clist.size() * sizeof(T),
    // hipMemcpyDeviceToHost));

    // CHECK_CUDA_ERR(utils::hipFree(d_clist, d_neighbors, d_neighborsCount, d_x, d_y, d_z, d_h, d_m, d_bbox, d_ro));
    CHECK_CUDA_ERR(utils::hipFree(d_clist, d_neighbors, d_neighborsCount));
}

template void computeDensity<double, ParticlesData<double>>(const std::vector<Task> &taskList, ParticlesData<double> &d);
template void computeDensity<double, ParticlesData<double>>(const Task &task, ParticlesData<double> &d);

template void copyInDensity<double, ParticlesData<double>>(ParticlesData<double> &d);
template void copyOutDensity<double, ParticlesData<double>>(ParticlesData<double> &d);
template <typename T, class Dataset>
void copyInDensity(Dataset &d)

{
    const size_t np = d.x.size();
    const size_t size_np_T = np * sizeof(T);
    const size_t size_bbox = sizeof(BBox<T>);

    CHECK_CUDA_ERR(utils::hipMalloc(size_np_T, d.d_x, d.d_y, d.d_z, d.d_h, d.d_m, d.d_ro));
    CHECK_CUDA_ERR(utils::hipMalloc(size_bbox, d.d_bbox));
    CHECK_CUDA_ERR(hipMemcpyAsync(d.d_x, d.x.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpyAsync(d.d_y, d.y.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpyAsync(d.d_z, d.z.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpyAsync(d.d_h, d.h.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpyAsync(d.d_m, d.m.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpyAsync(d.d_bbox, &d.bbox, size_bbox, hipMemcpyHostToDevice));
}

template <typename T, class Dataset>
void copyOutDensity(Dataset &d)
{
    const size_t np = d.x.size();
    const size_t size_np_T = np * sizeof(T);
    // CHECK_CUDA_ERR(hipMemcpyAsync(d.ro.data() + task.clist.front(), d.d_ro + task.clist.front(), task.clist.size() * sizeof(T),
    // hipMemcpyDeviceToHost));
    CHECK_CUDA_ERR(hipMemcpyAsync(d.ro.data(), d.d_ro, size_np_T, hipMemcpyDeviceToHost));

    CHECK_CUDA_ERR(utils::hipFree(d.d_x, d.d_y, d.d_z, d.d_h, d.d_m, d.d_bbox, d.d_ro));
}

#include <cstdlib>
#include <new>
#include <limits>
template <class T>
struct CudaAllocator
{
    typedef T value_type;
    CudaAllocator() = default;
    template <class U>
    constexpr CudaAllocator(const CudaAllocator<U> &) noexcept {}[[nodiscard]] T *allocate(std::size_t n)
    {
        if (n > std::numeric_limits<std::size_t>::max() / sizeof(T)) throw std::bad_alloc();
        if (auto p = static_cast<T *>(std::malloc(n * sizeof(T)))) return p;
        throw std::bad_alloc();
    }
    void deallocate(T *p, std::size_t) noexcept { std::free(p); }
};
template <class T, class U>
bool operator==(const CudaAllocator<T> &, const CudaAllocator<U> &)
{
    return true;
}
template <class T, class U>
bool operator!=(const CudaAllocator<T> &, const CudaAllocator<U> &)
{
    return false;
}

} // namespace cuda
} // namespace sph
} // namespace sphexa
