#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <algorithm>

#include "sph.cuh"
#include "BBox.hpp"
#include "ParticlesData.hpp"
#include "cudaUtils.cuh"
#include "../kernels.hpp"
#include "../lookupTables.hpp"

namespace sphexa
{
namespace sph
{
namespace cuda
{
namespace kernels
{
const double gradh_i = 1.0;
const double gradh_j = 1.0;

template <typename T>
__global__ void computeMomentumAndEnergyIAD(const int n, const T sincIndex, const T K, const int ngmax, const BBox<T> *bbox,
                                            const int *clist, const int *neighbors, const int *neighborsCount, const T *x, const T *y,
                                            const T *z, const T *vx, const T *vy, const T *vz, const T *h, const T *m, const T *ro,
                                            const T *p, const T *c, const T *c11, const T *c12, const T *c13, const T *c22, const T *c23,
                                            const T *c33, const T *wh, const T *whd, const size_t ltsize, T *grad_P_x, T *grad_P_y, T *grad_P_z, T *du, T *maxvsignal)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= n) return;

    const int i = clist[tid];
    const int nn = neighborsCount[tid];
    
    T maxvsignali = 0.0;
    T momentum_x = 0.0, momentum_y = 0.0, momentum_z = 0.0, energy = 0.0, energyAV = 0.0;
    for (int pj = 0; pj < nn; ++pj)
    {
        const int j = neighbors[tid * ngmax + pj];

        T r_ijx = (x[i] - x[j]);
        T r_ijy = (y[i] - y[j]);
        T r_ijz = (z[i] - z[j]);

        T r_jix = (x[j] - x[i]);
        T r_jiy = (y[j] - y[i]);
        T r_jiz = (z[j] - z[i]);

        applyPBC(*bbox, 2.0 * h[i], r_ijx, r_ijy, r_ijz);
        applyPBC(*bbox, 2.0 * h[i], r_jix, r_jiy, r_jiz);

        const T dist = std::sqrt(r_ijx * r_ijx + r_ijy * r_ijy + r_ijz * r_ijz);

        const T v_ijx = (vx[i] - vx[j]);
        const T v_ijy = (vy[i] - vy[j]);
        const T v_ijz = (vz[i] - vz[j]);

        const T v1 = dist / h[i];
        const T v2 = dist / h[j];

        const T rv = r_ijx * v_ijx + r_ijy * v_ijy + r_ijz * v_ijz;

        const T w1 = K * math_namespace::pow(lt::wharmonic_lt_with_derivative(wh, whd, ltsize, v1), (int)sincIndex);
        const T w2 = K * math_namespace::pow(lt::wharmonic_lt_with_derivative(wh, whd, ltsize, v2), (int)sincIndex);

        const T W1 = w1 / (h[i] * h[i] * h[i]);
        const T W2 = w2 / (h[j] * h[j] * h[j]);

        const T kern11_i = c11[i] * r_jix;
        const T kern12_i = c12[i] * r_jiy;
        const T kern13_i = c13[i] * r_jiz;
        const T kern21_i = c12[i] * r_jix;
        const T kern22_i = c22[i] * r_jiy;
        const T kern23_i = c23[i] * r_jiz;
        const T kern31_i = c13[i] * r_jix;
        const T kern32_i = c23[i] * r_jiy;
        const T kern33_i = c33[i] * r_jiz;

        const T kern11_j = c11[j] * r_jix;
        const T kern12_j = c12[j] * r_jiy;
        const T kern13_j = c13[j] * r_jiz;
        const T kern21_j = c12[j] * r_jix;
        const T kern22_j = c22[j] * r_jiy;
        const T kern23_j = c23[j] * r_jiz;
        const T kern31_j = c13[j] * r_jix;
        const T kern32_j = c23[j] * r_jiy;
        const T kern33_j = c33[j] * r_jiz;

        const T termA1_i = (kern11_i + kern12_i + kern13_i) * W1;
        const T termA2_i = (kern21_i + kern22_i + kern23_i) * W1;
        const T termA3_i = (kern31_i + kern32_i + kern33_i) * W1;

        const T termA1_j = (kern11_j + kern12_j + kern13_j) * W2;
        const T termA2_j = (kern21_j + kern22_j + kern23_j) * W2;
        const T termA3_j = (kern31_j + kern32_j + kern33_j) * W2;

        const T pro_i = p[i] / (gradh_i * ro[i] * ro[i]);
        const T pro_j = p[j] / (gradh_j * ro[j] * ro[j]);

        const T r_square = dist * dist;
        const T viscosity_ij = artificial_viscosity(ro[i], ro[j], h[i], h[j], c[i], c[j], rv, r_square);
        
        // For time-step calculations
        const T wij = rv / dist;
        const T vijsignal = c[i] + c[j] - 3.0 * wij;
        if (vijsignal > maxvsignali) maxvsignali = vijsignal;

        const T grad_Px_AV = 0.5 * (m[i] / ro[i] * viscosity_ij * termA1_i + m[j] / ro[j] * viscosity_ij * termA1_j);
        const T grad_Py_AV = 0.5 * (m[i] / ro[i] * viscosity_ij * termA2_i + m[j] / ro[j] * viscosity_ij * termA2_j);
        const T grad_Pz_AV = 0.5 * (m[i] / ro[i] * viscosity_ij * termA3_i + m[j] / ro[j] * viscosity_ij * termA3_j);

        momentum_x += m[j] * (pro_i * termA1_i + pro_j * termA1_j) + grad_Px_AV;
        momentum_y += m[j] * (pro_i * termA2_i + pro_j * termA2_j) + grad_Py_AV;
        momentum_z += m[j] * (pro_i * termA3_i + pro_j * termA3_j) + grad_Pz_AV;

        energy += m[j] * 2.0 * pro_i * (v_ijx * termA1_i + v_ijy * termA2_i + v_ijz * termA3_i);
        energyAV += grad_Px_AV * v_ijx + grad_Py_AV * v_ijy + grad_Pz_AV * v_ijz;
    }

    du[i] = 0.5 * (energy + energyAV);
    grad_P_x[i] = momentum_x;
    grad_P_y[i] = momentum_y;
    grad_P_z[i] = momentum_z;
    maxvsignal[i] = maxvsignali;
}

template <typename T>
__global__ void findNeighbors(const DeviceLinearOctree<T> o, const int *clist, const int n, const T *x, const T *y, const T *z, const T *h, const T displx,
                              const T disply, const T displz, const int max, const int may, const int maz, const int ngmax, int *neighbors, int *neighborsCount);

} // namespace kernels

template <typename T, class Dataset>
void computeMomentumAndEnergyIAD(const LinearOctree<T> &o, const std::vector<Task> &taskList, Dataset &d)
{
    const int maz = d.bbox.PBCz ? 2 : 0;
    const int may = d.bbox.PBCy ? 2 : 0;
    const int max = d.bbox.PBCx ? 2 : 0;
    
    const T displx = o.xmax[0] - o.xmin[0];
    const T disply = o.ymax[0] - o.ymin[0];
    const T displz = o.zmax[0] - o.zmin[0];

    const size_t np = d.x.size();
    const size_t size_np_T = np * sizeof(T);
    const T ngmax = taskList.empty() ? 0 : taskList.front().ngmax;

    const auto largestChunkSize =
        std::max_element(taskList.cbegin(), taskList.cend(),
                         [](const Task &lhs, const Task &rhs) { return lhs.clist.size() < rhs.clist.size(); })
            ->clist.size();

    const size_t size_largerNeighborsChunk_int = largestChunkSize * ngmax * sizeof(int);
    const size_t size_largerNChunk_int = largestChunkSize * sizeof(int);

    // number of streams to use
    const int NST = 3;

    // const size_t size_bbox = sizeof(BBox<T>);
    // const size_t size_np_T = np * sizeof(T);
    // const size_t size_n_int = n * sizeof(int);
    // const size_t size_n_T = n * sizeof(T);
    // const size_t size_allNeighbors = allNeighbors * sizeof(int);

    int *d_clist[NST], *d_neighbors[NST], *d_neighborsCount[NST]; // work arrays per stream

    const size_t ltsize = d.wh.size();

    // input data
    //CHECK_CUDA_ERR(utils::hipMalloc(size_np_T, d.devicePtrs.d_vx, d.devicePtrs.d_vy, d.devicePtrs.d_vz, d.devicePtrs.d_p, d.devicePtrs.d_c, d.devicePtrs.d_grad_P_x, d.devicePtrs.d_grad_P_y, d.devicePtrs.d_grad_P_z, d.devicePtrs.d_du, d.devicePtrs.d_maxvsignal));
    for (int i = 0; i < NST; ++i)
        CHECK_CUDA_ERR(utils::hipMalloc(size_largerNChunk_int, d_clist[i], d_neighborsCount[i]));
    for (int i = 0; i < NST; ++i)
        CHECK_CUDA_ERR(utils::hipMalloc(size_largerNeighborsChunk_int, d_neighbors[i]));

    CHECK_CUDA_ERR(hipMemcpy(d.devicePtrs.d_vx, d.vx.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d.devicePtrs.d_vy, d.vy.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d.devicePtrs.d_vz, d.vz.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d.devicePtrs.d_p, d.p.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d.devicePtrs.d_c, d.c.data(), size_np_T, hipMemcpyHostToDevice));

    CHECK_CUDA_ERR(hipMemcpy(d.devicePtrs.d_c11, d.c11.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d.devicePtrs.d_c12, d.c12.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d.devicePtrs.d_c13, d.c13.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d.devicePtrs.d_c22, d.c22.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d.devicePtrs.d_c23, d.c23.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d.devicePtrs.d_c33, d.c33.data(), size_np_T, hipMemcpyHostToDevice));

    hipStream_t streams[NST];
    for (int i = 0; i < NST; ++i)
        CHECK_CUDA_ERR(hipStreamCreate(&streams[i]));

    //DeviceLinearOctree<T> d_o;
    //d_o.mapLinearOctreeToDevice(o);
    
    for (int i = 0; i < taskList.size(); ++i)
    {
        const auto &t = taskList[i];

        const int sIdx = i % NST;
        hipStream_t stream = streams[sIdx];

        int *d_clist_use = d_clist[sIdx];
        int *d_neighbors_use = d_neighbors[sIdx];
        int *d_neighborsCount_use = d_neighborsCount[sIdx];

        const size_t n = t.clist.size();
        const size_t size_n_int = n * sizeof(int);
        // const size_t size_nNeighbors = n * ngmax * sizeof(int);

        CHECK_CUDA_ERR(hipMemcpyAsync(d_clist_use, t.clist.data(), size_n_int, hipMemcpyHostToDevice, stream));
        //CHECK_CUDA_ERR(hipMemcpyAsync(d_neighbors_use, t.neighbors.data(), size_nNeighbors, hipMemcpyHostToDevice, stream));
        //CHECK_CUDA_ERR(hipMemcpyAsync(d_neighborsCount_use, t.neighborsCount.data(), size_n_int, hipMemcpyHostToDevice, stream));

        const int threadsPerBlock = 256;
        const int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

        kernels::findNeighbors<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
            d.devicePtrs.d_o, d_clist_use, n, d.devicePtrs.d_x, d.devicePtrs.d_y, d.devicePtrs.d_z, d.devicePtrs.d_h, displx, disply, displz, max, may, maz, ngmax, d_neighbors_use, d_neighborsCount_use
        );

        kernels::computeMomentumAndEnergyIAD<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
            n, d.sincIndex, d.K, ngmax, d.devicePtrs.d_bbox, d_clist_use, d_neighbors_use, d_neighborsCount_use, d.devicePtrs.d_x, d.devicePtrs.d_y, d.devicePtrs.d_z, d.devicePtrs.d_vx, d.devicePtrs.d_vy, d.devicePtrs.d_vz, d.devicePtrs.d_h, d.devicePtrs.d_m, d.devicePtrs.d_ro,
            d.devicePtrs.d_p, d.devicePtrs.d_c, d.devicePtrs.d_c11, d.devicePtrs.d_c12, d.devicePtrs.d_c13, d.devicePtrs.d_c22, d.devicePtrs.d_c23, d.devicePtrs.d_c33, d.devicePtrs.d_wh, d.devicePtrs.d_whd, ltsize, d.devicePtrs.d_grad_P_x, d.devicePtrs.d_grad_P_y, d.devicePtrs.d_grad_P_z, d.devicePtrs.d_du, d.devicePtrs.d_maxvsignal);

        CHECK_CUDA_ERR(hipGetLastError());
    }

    d.devicePtrs.d_o.unmapLinearOctreeFromDevice();

    CHECK_CUDA_ERR(hipMemcpy(d.grad_P_x.data(), d.devicePtrs.d_grad_P_x, size_np_T, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERR(hipMemcpy(d.grad_P_y.data(), d.devicePtrs.d_grad_P_y, size_np_T, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERR(hipMemcpy(d.grad_P_z.data(), d.devicePtrs.d_grad_P_z, size_np_T, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERR(hipMemcpy(d.du.data(), d.devicePtrs.d_du, size_np_T, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERR(hipMemcpy(d.maxvsignal.data(), d.devicePtrs.d_maxvsignal, size_np_T, hipMemcpyDeviceToHost));

   for (int i = 0; i < NST; ++i)
        CHECK_CUDA_ERR(hipStreamDestroy(streams[i]));

    //CHECK_CUDA_ERR(utils::hipFree(d.devicePtrs.d_bbox, d.devicePtrs.d_x, d.devicePtrs.d_y, d.devicePtrs.d_z, d.devicePtrs.d_vx, d.devicePtrs.d_vy, d.devicePtrs.d_vz, d.devicePtrs.d_h, d.devicePtrs.d_m, d.devicePtrs.d_ro, d.devicePtrs.d_p,
    //    d.devicePtrs.d_c, d.devicePtrs.d_c11, d.devicePtrs.d_c12, d.devicePtrs.d_c13, d.devicePtrs.d_c22, d.devicePtrs.d_c23, d.devicePtrs.d_c33, d.devicePtrs.d_grad_P_x, d.devicePtrs.d_grad_P_y, d.devicePtrs.d_grad_P_z, d.devicePtrs.d_du, d.devicePtrs.d_maxvsignal, d.devicePtrs.d_wh, d.devicePtrs.d_whd));
    for (int i = 0; i < NST; ++i)
        CHECK_CUDA_ERR(utils::hipFree(d_clist[i], d_neighbors[i], d_neighborsCount[i]));
}

template void computeMomentumAndEnergyIAD<double, ParticlesData<double>>(const LinearOctree<double> &o, const std::vector<Task> &taskList, ParticlesData<double> &d);

} // namespace cuda
} // namespace sph
} // namespace sphexa
