#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <algorithm>

#include "sph.cuh"
#include "BBox.hpp"
#include "ParticlesData.hpp"
#include "cudaUtils.cuh"
#include "../kernels.hpp"
#include "../lookupTables.hpp"

namespace sphexa
{
namespace sph
{
namespace cuda
{
namespace kernels
{
template <typename T>
__global__ void computeIAD(const int n, const T sincIndex, const T K, const int ngmax, const BBox<T> *bbox, const int *clist,
                           const int *neighbors, const int *neighborsCount, const T *x, const T *y, const T *z, const T *h, const T *m, 
                           const T *ro, const T *wh, const T *whd, const size_t ltsize, T *c11, T *c12, T *c13, T *c22, T *c23, T *c33)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= n) return;

    const int i = clist[tid];
    const int nn = neighborsCount[tid];

    T tau11 = 0.0, tau12 = 0.0, tau13 = 0.0, tau22 = 0.0, tau23 = 0.0, tau33 = 0.0;
    for (int pj = 0; pj < nn; ++pj)
    {
        const int j = neighbors[tid * ngmax + pj];

        const T dist = distancePBC(*bbox, h[i], x[i], y[i], z[i], x[j], y[j], z[j]);
        const T vloc = dist / h[i];

        const T w = K * math_namespace::pow(lt::wharmonic_lt_with_derivative(wh, whd, ltsize, vloc), (int)sincIndex);
        const T W = w / (h[i] * h[i] * h[i]);

        T r_ijx = (x[i] - x[j]);
        T r_ijy = (y[i] - y[j]);
        T r_ijz = (z[i] - z[j]);

        applyPBC(*bbox, 2.0 * h[i], r_ijx, r_ijy, r_ijz);

        tau11 += r_ijx * r_ijx * m[j] / ro[j] * W;
        tau12 += r_ijx * r_ijy * m[j] / ro[j] * W;
        tau13 += r_ijx * r_ijz * m[j] / ro[j] * W;
        tau22 += r_ijy * r_ijy * m[j] / ro[j] * W;
        tau23 += r_ijy * r_ijz * m[j] / ro[j] * W;
        tau33 += r_ijz * r_ijz * m[j] / ro[j] * W;
    }

    const T det =
        tau11 * tau22 * tau33 + 2.0 * tau12 * tau23 * tau13 - tau11 * tau23 * tau23 - tau22 * tau13 * tau13 - tau33 * tau12 * tau12;

    c11[i] = (tau22 * tau33 - tau23 * tau23) / det;
    c12[i] = (tau13 * tau23 - tau33 * tau12) / det;
    c13[i] = (tau12 * tau23 - tau22 * tau13) / det;
    c22[i] = (tau11 * tau33 - tau13 * tau13) / det;
    c23[i] = (tau13 * tau12 - tau11 * tau23) / det;
    c33[i] = (tau11 * tau22 - tau12 * tau12) / det;
}

template <typename T>
__global__ void findNeighbors(const DeviceLinearOctree<T> o, const int *clist, const int n, const T *x, const T *y, const T *z, const T *h, const T displx,
                              const T disply, const T displz, const int max, const int may, const int maz, const int ngmax, int *neighbors, int *neighborsCount);

} // namespace kernels

template <typename T, class Dataset>
void computeIAD(const LinearOctree<T> &o, const std::vector<Task> &taskList, Dataset &d)
{
    const int maz = d.bbox.PBCz ? 2 : 0;
    const int may = d.bbox.PBCy ? 2 : 0;
    const int max = d.bbox.PBCx ? 2 : 0;
    
    const T displx = o.xmax[0] - o.xmin[0];
    const T disply = o.ymax[0] - o.ymin[0];
    const T displz = o.zmax[0] - o.zmin[0];

    const size_t np = d.x.size();
    const size_t size_np_T = np * sizeof(T);
    const T ngmax = taskList.empty() ? 0 : taskList.front().ngmax;

    const auto largestChunkSize =
        std::max_element(taskList.cbegin(), taskList.cend(),
                         [](const Task &lhs, const Task &rhs) { return lhs.clist.size() < rhs.clist.size(); })
            ->clist.size();

    const size_t size_largerNeighborsChunk_int = largestChunkSize * ngmax * sizeof(int);
    const size_t size_largerNChunk_int = largestChunkSize * sizeof(int);

    // number of CUDA streams to use
    const int NST = 2;

    // device pointers - d_ prefix stands for device
    int *d_clist[NST], *d_neighbors[NST], *d_neighborsCount[NST]; // work arrays per stream

    const size_t ltsize = d.wh.size();

    // input data
    CHECK_CUDA_ERR(utils::hipMalloc(size_np_T, d.devicePtrs.d_c11, d.devicePtrs.d_c12, d.devicePtrs.d_c13, d.devicePtrs.d_c22, d.devicePtrs.d_c23, d.devicePtrs.d_c33));

    for (int i = 0; i < NST; ++i)
        CHECK_CUDA_ERR(utils::hipMalloc(size_largerNChunk_int, d_clist[i], d_neighborsCount[i]));
    for (int i = 0; i < NST; ++i)
        CHECK_CUDA_ERR(utils::hipMalloc(size_largerNeighborsChunk_int, d_neighbors[i]));

    
    // CHECK_CUDA_ERR(hipMemcpy(d_x, d.x.data(), size_np_T, hipMemcpyHostToDevice));
    // CHECK_CUDA_ERR(hipMemcpy(d_y, d.y.data(), size_np_T, hipMemcpyHostToDevice));
    // CHECK_CUDA_ERR(hipMemcpy(d_z, d.z.data(), size_np_T, hipMemcpyHostToDevice));
    // CHECK_CUDA_ERR(hipMemcpy(d_h, d.h.data(), size_np_T, hipMemcpyHostToDevice));
    // CHECK_CUDA_ERR(hipMemcpy(d_m, d.m.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d.devicePtrs.d_ro, d.ro.data(), size_np_T, hipMemcpyHostToDevice));
    // CHECK_CUDA_ERR(hipMemcpy(d_wh, d.wh.data(), size_lt_T, hipMemcpyHostToDevice));
    // CHECK_CUDA_ERR(hipMemcpy(d_whd, d.whd.data(), size_lt_T, hipMemcpyHostToDevice));
    // CHECK_CUDA_ERR(hipMemcpy(d_bbox, &d.bbox, size_bbox, hipMemcpyHostToDevice));
    

    hipStream_t streams[NST];
    for (int i = 0; i < NST; ++i)
        CHECK_CUDA_ERR(hipStreamCreate(&streams[i]));

    //DeviceLinearOctree<T> d_o;
    //d.devicePtrs.d_o.mapLinearOctreeToDevice(o);
    
    for (int i = 0; i < taskList.size(); ++i)
    {
        const auto &t = taskList[i];

        const int sIdx = i % NST;
        hipStream_t stream = streams[sIdx];

        int *d_clist_use = d_clist[sIdx];
        int *d_neighbors_use = d_neighbors[sIdx];
        int *d_neighborsCount_use = d_neighborsCount[sIdx];

        const size_t n = t.clist.size();
        const size_t size_n_int = n * sizeof(int);
        // const size_t size_nNeighbors = n * ngmax * sizeof(int);

        CHECK_CUDA_ERR(hipMemcpyAsync(d_clist_use, t.clist.data(), size_n_int, hipMemcpyHostToDevice, stream));
        //CHECK_CUDA_ERR(hipMemcpyAsync(d_neighbors_use, t.neighbors.data(), size_nNeighbors, hipMemcpyHostToDevice, stream));
        //CHECK_CUDA_ERR(hipMemcpyAsync(d_neighborsCount_use, t.neighborsCount.data(), size_n_int, hipMemcpyHostToDevice, stream));

        const int threadsPerBlock = 256;
        const int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

        kernels::findNeighbors<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
            d.devicePtrs.d_o, d_clist_use, n, d.devicePtrs.d_x, d.devicePtrs.d_y, d.devicePtrs.d_z, d.devicePtrs.d_h, displx, disply, displz, max, may, maz, ngmax, d_neighbors_use, d_neighborsCount_use
        );

        // printf("CUDA IAD kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

        kernels::computeIAD<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(n, d.sincIndex, d.K, ngmax, d.devicePtrs.d_bbox, d_clist_use, d_neighbors_use,
            d_neighborsCount_use, d.devicePtrs.d_x, d.devicePtrs.d_y, d.devicePtrs.d_z, d.devicePtrs.d_h, d.devicePtrs.d_m, d.devicePtrs.d_ro, d.devicePtrs.d_wh, d.devicePtrs.d_whd, ltsize, d.devicePtrs.d_c11, d.devicePtrs.d_c12, d.devicePtrs.d_c13, d.devicePtrs.d_c22,
            d.devicePtrs.d_c23, d.devicePtrs.d_c33);
        CHECK_CUDA_ERR(hipGetLastError());
    }

    
    // d.d_o.unmapLinearOctreeFromDevice();

    CHECK_CUDA_ERR(hipMemcpy(d.c11.data(), d.devicePtrs.d_c11, size_np_T, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERR(hipMemcpy(d.c12.data(), d.devicePtrs.d_c12, size_np_T, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERR(hipMemcpy(d.c13.data(), d.devicePtrs.d_c13, size_np_T, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERR(hipMemcpy(d.c22.data(), d.devicePtrs.d_c22, size_np_T, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERR(hipMemcpy(d.c23.data(), d.devicePtrs.d_c23, size_np_T, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERR(hipMemcpy(d.c33.data(), d.devicePtrs.d_c33, size_np_T, hipMemcpyDeviceToHost));
    

    for (int i = 0; i < NST; ++i)
        CHECK_CUDA_ERR(hipStreamDestroy(streams[i]));

    for (int i = 0; i < NST; ++i)
        CHECK_CUDA_ERR(utils::hipFree(d_clist[i], d_neighbors[i], d_neighborsCount[i]));
}

template void computeIAD<double, ParticlesData<double>>(const LinearOctree<double> &o, const std::vector<Task> &taskList, ParticlesData<double> &d);

} // namespace cuda
} // namespace sph
} // namespace sphexa
