#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <algorithm>

#include "sph.cuh"
#include "BBox.hpp"
#include "ParticlesData.hpp"
#include "cudaUtils.cuh"
#include "../kernels.hpp"
#include "../kernel/computeFindNeighbors.hpp"
#include "../kernel/computeIAD.hpp"

namespace sphexa
{
namespace sph
{
namespace cuda
{
template <typename T>
__global__ void computeIAD(const int n, const T sincIndex, const T K, const int ngmax, const BBox<T> *bbox, const int *clist,
                           const int *neighbors, const int *neighborsCount, const T *x, const T *y, const T *z, const T *h, const T *m, 
                           const T *ro, const T *wh, const T *whd, const size_t ltsize, T *c11, T *c12, T *c13, T *c22, T *c23, T *c33)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= n) return;

    sph::kernels::IADJLoop(tid, sincIndex, K, ngmax, bbox, clist, neighbors, neighborsCount, x, y, z, h, m, ro, wh, whd, ltsize, c11, c12, c13, c22, c23, c33);
}

template <typename T, class Dataset>
void computeIAD(const std::vector<Task> &taskList, Dataset &d)
{
    const int maz = d.bbox.PBCz ? 2 : 0;
    const int may = d.bbox.PBCy ? 2 : 0;
    const int max = d.bbox.PBCx ? 2 : 0;

    const T displx = d.devPtrs.d_o.xmax0 - d.devPtrs.d_o.xmin0;
    const T disply = d.devPtrs.d_o.ymax0 - d.devPtrs.d_o.ymin0;
    const T displz = d.devPtrs.d_o.zmax0 - d.devPtrs.d_o.zmin0;

    const size_t np = d.x.size();
    const size_t size_np_T = np * sizeof(T);
    const T ngmax = taskList.empty() ? 0 : taskList.front().ngmax;

    const size_t ltsize = d.wh.size();

    const auto largestChunkSize =
        std::max_element(taskList.cbegin(), taskList.cend(),
                         [](const Task &lhs, const Task &rhs) { return lhs.clist.size() < rhs.clist.size(); })
            ->clist.size();

    d.devPtrs.resize_streams(largestChunkSize, ngmax);

    // number of CUDA streams to use
    const int NST = DeviceParticlesData<T, Dataset>::NST;

    /*
    // device pointers - d_ prefix stands for device
    int *d_clist[NST], *d_neighbors[NST], *d_neighborsCount[NST]; // work arrays per stream

    // input data
    //CHECK_CUDA_ERR(utils::hipMalloc(size_np_T, d.d_c11, d.d_c12, d.d_c13, d.d_c22, d.d_c23, d.d_c33));

    for (int i = 0; i < NST; ++i)
        CHECK_CUDA_ERR(utils::hipMalloc(size_largerNChunk_int, d_clist[i], d_neighborsCount[i]));
    for (int i = 0; i < NST; ++i)
        CHECK_CUDA_ERR(utils::hipMalloc(size_largerNeighborsChunk_int, d_neighbors[i]));

    hipStream_t streams[NST];
    for (int i = 0; i < NST; ++i)
        CHECK_CUDA_ERR(hipStreamCreate(&streams[i]));
    */
    
    // CHECK_CUDA_ERR(hipMemcpy(d_x, d.x.data(), size_np_T, hipMemcpyHostToDevice));
    // CHECK_CUDA_ERR(hipMemcpy(d_y, d.y.data(), size_np_T, hipMemcpyHostToDevice));
    // CHECK_CUDA_ERR(hipMemcpy(d_z, d.z.data(), size_np_T, hipMemcpyHostToDevice));
    // CHECK_CUDA_ERR(hipMemcpy(d_h, d.h.data(), size_np_T, hipMemcpyHostToDevice));
    // CHECK_CUDA_ERR(hipMemcpy(d_m, d.m.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d.devPtrs.d_ro, d.ro.data(), size_np_T, hipMemcpyHostToDevice));
    // CHECK_CUDA_ERR(hipMemcpy(d_wh, d.wh.data(), size_lt_T, hipMemcpyHostToDevice));
    // CHECK_CUDA_ERR(hipMemcpy(d_whd, d.whd.data(), size_lt_T, hipMemcpyHostToDevice));
    // CHECK_CUDA_ERR(hipMemcpy(d_bbox, &d.bbox, size_bbox, hipMemcpyHostToDevice));

    //DeviceLinearOctree<T> d_o;
    //d.d_o.mapLinearOctreeToDevice(o);
    
    for (int i = 0; i < taskList.size(); ++i)
    {
        const auto &t = taskList[i];

        const int sIdx = i % NST;
        /*
        hipStream_t stream = streams[sIdx];

        int *d_clist_use = d_clist[sIdx];
        int *d_neighbors_use = d_neighbors[sIdx];
        int *d_neighborsCount_use = d_neighborsCount[sIdx];
        */
        hipStream_t stream = d.devPtrs.d_stream[sIdx].stream;

        int *d_clist_use = d.devPtrs.d_stream[sIdx].d_clist;
        int *d_neighbors_use = d.devPtrs.d_stream[sIdx].d_neighbors;
        int *d_neighborsCount_use = d.devPtrs.d_stream[sIdx].d_neighborsCount;

        const size_t n = t.clist.size();
        const size_t size_n_int = n * sizeof(int);
        // const size_t size_nNeighbors = n * ngmax * sizeof(int);

        CHECK_CUDA_ERR(hipMemcpyAsync(d_clist_use, t.clist.data(), size_n_int, hipMemcpyHostToDevice, stream));
        //CHECK_CUDA_ERR(hipMemcpyAsync(d_neighbors_use, t.neighbors.data(), size_nNeighbors, hipMemcpyHostToDevice, stream));
        //CHECK_CUDA_ERR(hipMemcpyAsync(d_neighborsCount_use, t.neighborsCount.data(), size_n_int, hipMemcpyHostToDevice, stream));

        const int threadsPerBlock = 256;
        const int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

        findNeighbors<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
            d.devPtrs.d_o, d_clist_use, n, d.devPtrs.d_x, d.devPtrs.d_y, d.devPtrs.d_z, d.devPtrs.d_h, displx, disply, displz, max, may, maz, ngmax, d_neighbors_use, d_neighborsCount_use
        );

        // printf("CUDA IAD kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

        computeIAD<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(n, d.sincIndex, d.K, ngmax, d.devPtrs.d_bbox, d_clist_use, d_neighbors_use,
            d_neighborsCount_use, d.devPtrs.d_x, d.devPtrs.d_y, d.devPtrs.d_z, d.devPtrs.d_h, d.devPtrs.d_m, d.devPtrs.d_ro, d.devPtrs.d_wh, d.devPtrs.d_whd, ltsize, d.devPtrs.d_c11, d.devPtrs.d_c12, d.devPtrs.d_c13, d.devPtrs.d_c22,
            d.devPtrs.d_c23, d.devPtrs.d_c33);
        CHECK_CUDA_ERR(hipGetLastError());
    }

    
    // d.d_o.unmapLinearOctreeFromDevice();

    CHECK_CUDA_ERR(hipMemcpy(d.c11.data(), d.devPtrs.d_c11, size_np_T, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERR(hipMemcpy(d.c12.data(), d.devPtrs.d_c12, size_np_T, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERR(hipMemcpy(d.c13.data(), d.devPtrs.d_c13, size_np_T, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERR(hipMemcpy(d.c22.data(), d.devPtrs.d_c22, size_np_T, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERR(hipMemcpy(d.c23.data(), d.devPtrs.d_c23, size_np_T, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERR(hipMemcpy(d.c33.data(), d.devPtrs.d_c33, size_np_T, hipMemcpyDeviceToHost));
    
    /*
    for (int i = 0; i < NST; ++i)
        CHECK_CUDA_ERR(hipStreamDestroy(streams[i]));

    for (int i = 0; i < NST; ++i)
        CHECK_CUDA_ERR(utils::hipFree(d_clist[i], d_neighbors[i], d_neighborsCount[i]));
    */
}

template void computeIAD<double, ParticlesData<double>>(const std::vector<Task> &taskList, ParticlesData<double> &d);

} // namespace cuda
} // namespace sph
} // namespace sphexa
