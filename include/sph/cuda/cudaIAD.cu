#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <algorithm>

#include "sph.cuh"
#include "BBox.hpp"
#include "ParticlesData.hpp"
#include "cudaUtils.cuh"
#include "../kernels.hpp"
#include "../lookupTables.hpp"

namespace sphexa
{
namespace sph
{
namespace cuda
{
namespace kernels
{
template <typename T>
__global__ void computeIAD(const int n, const T sincIndex, const T K, const int ngmax, const BBox<T> *bbox, const int *clist,
                           const int *neighbors, const int *neighborsCount, const T *x, const T *y, const T *z, const T *h, const T *m, 
                           const T *ro, const T *wh, const T *whd, const size_t ltsize, T *c11, T *c12, T *c13, T *c22, T *c23, T *c33)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= n) return;

    const int i = clist[tid];
    const int nn = neighborsCount[tid];

    T tau11 = 0.0, tau12 = 0.0, tau13 = 0.0, tau22 = 0.0, tau23 = 0.0, tau33 = 0.0;
    for (int pj = 0; pj < nn; ++pj)
    {
        const int j = neighbors[tid * ngmax + pj];

        const T dist = distancePBC(*bbox, h[i], x[i], y[i], z[i], x[j], y[j], z[j]);
        const T vloc = dist / h[i];

        const T w = K * math_namespace::pow(lt::wharmonic_lt_with_derivative(wh, whd, ltsize, vloc), (int)sincIndex);
        const T W = w / (h[i] * h[i] * h[i]);

        T r_ijx = (x[i] - x[j]);
        T r_ijy = (y[i] - y[j]);
        T r_ijz = (z[i] - z[j]);

        applyPBC(*bbox, 2.0 * h[i], r_ijx, r_ijy, r_ijz);

        tau11 += r_ijx * r_ijx * m[j] / ro[j] * W;
        tau12 += r_ijx * r_ijy * m[j] / ro[j] * W;
        tau13 += r_ijx * r_ijz * m[j] / ro[j] * W;
        tau22 += r_ijy * r_ijy * m[j] / ro[j] * W;
        tau23 += r_ijy * r_ijz * m[j] / ro[j] * W;
        tau33 += r_ijz * r_ijz * m[j] / ro[j] * W;
    }

    const T det =
        tau11 * tau22 * tau33 + 2.0 * tau12 * tau23 * tau13 - tau11 * tau23 * tau23 - tau22 * tau13 * tau13 - tau33 * tau12 * tau12;

    c11[i] = (tau22 * tau33 - tau23 * tau23) / det;
    c12[i] = (tau13 * tau23 - tau33 * tau12) / det;
    c13[i] = (tau12 * tau23 - tau22 * tau13) / det;
    c22[i] = (tau11 * tau33 - tau13 * tau13) / det;
    c23[i] = (tau13 * tau12 - tau11 * tau23) / det;
    c33[i] = (tau11 * tau22 - tau12 * tau12) / det;
}
} // namespace kernels

template <typename T, class Dataset>
void computeIAD(const std::vector<Task> &taskList, Dataset &d)
{
    const int maz = d.bbox.PBCz ? 2 : 0;
    const int may = d.bbox.PBCy ? 2 : 0;
    const int max = d.bbox.PBCx ? 2 : 0;

    const T displx = d.devPtrs.d_o.xmax0 - d.devPtrs.d_o.xmin0;
    const T disply = d.devPtrs.d_o.ymax0 - d.devPtrs.d_o.ymin0;
    const T displz = d.devPtrs.d_o.zmax0 - d.devPtrs.d_o.zmin0;

    const size_t np = d.x.size();
    const size_t size_np_T = np * sizeof(T);
    const T ngmax = taskList.empty() ? 0 : taskList.front().ngmax;

    const size_t ltsize = d.wh.size();

    const auto largestChunkSize =
        std::max_element(taskList.cbegin(), taskList.cend(),
                         [](const Task &lhs, const Task &rhs) { return lhs.clist.size() < rhs.clist.size(); })
            ->clist.size();

    d.devPtrs.resize_streams(largestChunkSize, ngmax);

    // number of CUDA streams to use
    const int NST = DeviceParticlesData<T, Dataset>::NST;

    /*
    // device pointers - d_ prefix stands for device
    int *d_clist[NST], *d_neighbors[NST], *d_neighborsCount[NST]; // work arrays per stream

    // input data
    //CHECK_CUDA_ERR(utils::hipMalloc(size_np_T, d.d_c11, d.d_c12, d.d_c13, d.d_c22, d.d_c23, d.d_c33));

    for (int i = 0; i < NST; ++i)
        CHECK_CUDA_ERR(utils::hipMalloc(size_largerNChunk_int, d_clist[i], d_neighborsCount[i]));
    for (int i = 0; i < NST; ++i)
        CHECK_CUDA_ERR(utils::hipMalloc(size_largerNeighborsChunk_int, d_neighbors[i]));

    hipStream_t streams[NST];
    for (int i = 0; i < NST; ++i)
        CHECK_CUDA_ERR(hipStreamCreate(&streams[i]));
    */
    
    // CHECK_CUDA_ERR(hipMemcpy(d_x, d.x.data(), size_np_T, hipMemcpyHostToDevice));
    // CHECK_CUDA_ERR(hipMemcpy(d_y, d.y.data(), size_np_T, hipMemcpyHostToDevice));
    // CHECK_CUDA_ERR(hipMemcpy(d_z, d.z.data(), size_np_T, hipMemcpyHostToDevice));
    // CHECK_CUDA_ERR(hipMemcpy(d_h, d.h.data(), size_np_T, hipMemcpyHostToDevice));
    // CHECK_CUDA_ERR(hipMemcpy(d_m, d.m.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d.devPtrs.d_ro, d.ro.data(), size_np_T, hipMemcpyHostToDevice));
    // CHECK_CUDA_ERR(hipMemcpy(d_wh, d.wh.data(), size_lt_T, hipMemcpyHostToDevice));
    // CHECK_CUDA_ERR(hipMemcpy(d_whd, d.whd.data(), size_lt_T, hipMemcpyHostToDevice));
    // CHECK_CUDA_ERR(hipMemcpy(d_bbox, &d.bbox, size_bbox, hipMemcpyHostToDevice));

    //DeviceLinearOctree<T> d_o;
    //d.d_o.mapLinearOctreeToDevice(o);
    
    for (int i = 0; i < taskList.size(); ++i)
    {
        const auto &t = taskList[i];

        const int sIdx = i % NST;
        /*
        hipStream_t stream = streams[sIdx];

        int *d_clist_use = d_clist[sIdx];
        int *d_neighbors_use = d_neighbors[sIdx];
        int *d_neighborsCount_use = d_neighborsCount[sIdx];
        */
        hipStream_t stream = d.devPtrs.d_stream[sIdx].stream;

        int *d_clist_use = d.devPtrs.d_stream[sIdx].d_clist;
        int *d_neighbors_use = d.devPtrs.d_stream[sIdx].d_neighbors;
        int *d_neighborsCount_use = d.devPtrs.d_stream[sIdx].d_neighborsCount;

        const size_t n = t.clist.size();
        const size_t size_n_int = n * sizeof(int);
        // const size_t size_nNeighbors = n * ngmax * sizeof(int);

        CHECK_CUDA_ERR(hipMemcpyAsync(d_clist_use, t.clist.data(), size_n_int, hipMemcpyHostToDevice, stream));
        //CHECK_CUDA_ERR(hipMemcpyAsync(d_neighbors_use, t.neighbors.data(), size_nNeighbors, hipMemcpyHostToDevice, stream));
        //CHECK_CUDA_ERR(hipMemcpyAsync(d_neighborsCount_use, t.neighborsCount.data(), size_n_int, hipMemcpyHostToDevice, stream));

        const int threadsPerBlock = 256;
        const int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

        kernels::findNeighbors<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
            d.devPtrs.d_o, d_clist_use, n, d.devPtrs.d_x, d.devPtrs.d_y, d.devPtrs.d_z, d.devPtrs.d_h, displx, disply, displz, max, may, maz, ngmax, d_neighbors_use, d_neighborsCount_use
        );

        // printf("CUDA IAD kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

        kernels::computeIAD<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(n, d.sincIndex, d.K, ngmax, d.devPtrs.d_bbox, d_clist_use, d_neighbors_use,
            d_neighborsCount_use, d.devPtrs.d_x, d.devPtrs.d_y, d.devPtrs.d_z, d.devPtrs.d_h, d.devPtrs.d_m, d.devPtrs.d_ro, d.devPtrs.d_wh, d.devPtrs.d_whd, ltsize, d.devPtrs.d_c11, d.devPtrs.d_c12, d.devPtrs.d_c13, d.devPtrs.d_c22,
            d.devPtrs.d_c23, d.devPtrs.d_c33);
        CHECK_CUDA_ERR(hipGetLastError());
    }

    
    // d.d_o.unmapLinearOctreeFromDevice();

    CHECK_CUDA_ERR(hipMemcpy(d.c11.data(), d.devPtrs.d_c11, size_np_T, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERR(hipMemcpy(d.c12.data(), d.devPtrs.d_c12, size_np_T, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERR(hipMemcpy(d.c13.data(), d.devPtrs.d_c13, size_np_T, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERR(hipMemcpy(d.c22.data(), d.devPtrs.d_c22, size_np_T, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERR(hipMemcpy(d.c23.data(), d.devPtrs.d_c23, size_np_T, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERR(hipMemcpy(d.c33.data(), d.devPtrs.d_c33, size_np_T, hipMemcpyDeviceToHost));
    
    /*
    for (int i = 0; i < NST; ++i)
        CHECK_CUDA_ERR(hipStreamDestroy(streams[i]));

    for (int i = 0; i < NST; ++i)
        CHECK_CUDA_ERR(utils::hipFree(d_clist[i], d_neighbors[i], d_neighborsCount[i]));
    */
}

template void computeIAD<double, ParticlesData<double>>(const std::vector<Task> &taskList, ParticlesData<double> &d);

} // namespace cuda
} // namespace sph
} // namespace sphexa
